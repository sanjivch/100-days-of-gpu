__global__ void vecadd( const float* A, const float* B, float* C, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x ;

    if (i < n){
        C[i] = A[i] + 2.5*B[i];
    }
}

#include <hip/hip_runtime.h>
#include<iostream>
int main(){
    const int n = 1024;
    const int size = n*sizeof(float);

    float h_A[n], h_B[n], h_C[n];
    float *d_A, *d_B, *d_C;

    // h_A = new float [n];
    // h_B = new float [n];
    // h_C = new float [n];

    for(int i = 0 ;i <n;i++){
        h_A[i] = 1.0;
        h_B[i] = 0.5*i;
        
    }

    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);

    hipMemcpy(d_A, h_A, size,hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B,size,hipMemcpyHostToDevice);

     int threadsPerBlock = 256;
    int blocksPerGrid = ( n + threadsPerBlock -1) / threadsPerBlock;

    vecadd<<<blocksPerGrid, threadsPerBlock>>>(d_A,d_B,d_C,n);

    hipMemcpy(h_C,d_C, size,hipMemcpyDeviceToHost);
    for(int i =0;i<n;i++){
        printf("%d\t%f\t%f\t%f\n", i, d_A[i], d_B[i], d_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


}
