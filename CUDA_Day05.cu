
#include <hip/hip_runtime.h>
#include<iostream>


__global__ void printThreadIdx(int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) { // Ensure the thread is within bounds
        printf("Block: %d, Thread: %d, block dim: %d, Global Index: %d\n", blockIdx.x, threadIdx.x, blockDim.x, idx);
    }
}

int main(){

    printf("Hello world\n");

    // 

    printThreadIdx<<<20, 32>>>(604);

    hipDeviceSynchronize();
}
