#include <stdio.h>
#include <hip/hip_runtime.h>

// Convert RGB to grayscale on GPU
__global__ void convertToGray(unsigned char* rgb, unsigned char* gray, int width, int height) {
    // Get pixel position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    
    if (x < width && y < height) {
        // Calculate position in RGB array (3 values per pixel)
        int pos = (y * width + x) * 3;
        
        // Get RGB values
        int r = rgb[pos];
        int g = rgb[pos + 1];
        int b = rgb[pos + 2];
        
        // Simple average of RGB values
        gray[y * width + x] = (r + g + b) / 3;
    }
}

int main() {
    // Image size
    int width = 640;
    int height = 480;
    
    // Allocate memory for input RGB image
    unsigned char* inputImage = new unsigned char[width * height * 3];
    
    // Fill with white pixels for testing
    for (int i = 0; i < width * height * 3; i++) {
        inputImage[i] = 255;
    }
    
    // Allocate memory for output grayscale image
    unsigned char* outputImage = new unsigned char[width * height];
    
    // GPU memory pointers
    unsigned char *d_input, *d_output;
    
    // Allocate GPU memory
    hipMalloc(&d_input, width * height * 3);
    hipMalloc(&d_output, width * height);
    
    // Copy input image to GPU
    hipMemcpy(d_input, inputImage, width * height * 3, hipMemcpyHostToDevice);
    
    // Set up grid and blocks
    dim3 block(16, 16);  // 16x16 threads per block
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    
    // Run conversion on GPU
    convertToGray<<<grid, block>>>(d_input, d_output, width, height);
    
    // Copy result back to CPU
    hipMemcpy(outputImage, d_output, width * height, hipMemcpyDeviceToHost);
    
    // Clean up
    hipFree(d_input);
    hipFree(d_output);

}
